#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/vision_layers.hpp"

namespace caffe {

template <typename Dtype>
void ConvolutionLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {
  const Dtype* weight = this->blobs_[0]->gpu_data();
  for (int i = 0; i < bottom.size(); ++i) {
    const Dtype* bottom_data = bottom[i]->gpu_data();
    Dtype* top_data = top[i]->mutable_gpu_data();
    for (int n = 0; n < this->num_; ++n) {
      this->forward_gpu_gemm(bottom_data + n * this->bottom_dim_, weight,
          top_data + n * this->top_dim_);
      if (this->bias_term_) {
        const Dtype* bias = this->blobs_[1]->gpu_data();
        this->forward_gpu_bias(top_data + n * this->top_dim_, bias);
      }
    }
  }
//  if (CheckForNanGPU<Dtype>(top[0]->count(), top[0]->gpu_data()))
//  {
//    std::stringstream ss;
//    ss << "Found nan in conv layer top: " << this->layer_param_.name() << std::endl;
//    for(int i=0; i < this->blobs_.size(); ++i)
//    {
//      Blob<Dtype>* weight_blob = this->blobs_[i].get();
//      if (CheckForNanGPU(weight_blob->count(), weight_blob->gpu_data()))
//      {
//        ss << "Param blob had nan" << std::endl;
//      } else {
//        ss << "Param blob did not have nan" << std::endl;
//      }
//    }

//    {
//      std::fstream of;
//      of.open("nan_blob_o.csv");
//      PrintBlob(of, "BlobWithNan",*top[0]);
//    }

//    {
//      std::ofstream of;
//      of.open("nan_blob_input.csv");
//      PrintBlob(of, "BlobWithNan",*bottom[0]);
//    }

//    {
//      std::ofstream of;
//      of.open("nan_blob_weights.csv");
//      PrintBlob(of, "BlobWithNan",*(this->blobs_[0]));
//    }

//    {
//      std::ofstream of;
//      of.open("nan_blob_biases.csv");
//      PrintBlob(of, "BlobWithNan",*(this->blobs_[1]));
//    }

//    LOG(ERROR) << ss.str();
//  }
}

template <typename Dtype>
void ConvolutionLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  const Dtype* weight = this->blobs_[0]->gpu_data();
  Dtype* weight_diff = this->blobs_[0]->mutable_gpu_diff();
  for (int i = 0; i < top.size(); ++i) {
    const Dtype* top_diff = top[i]->gpu_diff();
    // Bias gradient, if necessary.
    if (this->bias_term_ && this->param_propagate_down_[1]) {
      Dtype* bias_diff = this->blobs_[1]->mutable_gpu_diff();
      for (int n = 0; n < this->num_; ++n) {
        this->backward_gpu_bias(bias_diff, top_diff + n * this->top_dim_);
      }
    }
    if (this->param_propagate_down_[0] || propagate_down[i]) {
      const Dtype* bottom_data = bottom[i]->gpu_data();
      Dtype* bottom_diff = bottom[i]->mutable_gpu_diff();
      for (int n = 0; n < this->num_; ++n) {
        // gradient w.r.t. weight. Note that we will accumulate diffs.
        if (this->param_propagate_down_[0]) {
          this->weight_gpu_gemm(bottom_data + n * this->bottom_dim_,
              top_diff + n * this->top_dim_, weight_diff);
        }
        // gradient w.r.t. bottom data, if necessary.
        if (propagate_down[i]) {
          this->backward_gpu_gemm(top_diff + n * this->top_dim_, weight,
              bottom_diff + n * this->bottom_dim_);
        }
      }
    }
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(ConvolutionLayer);

}  // namespace caffe
