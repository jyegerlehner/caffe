#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

#include "thrust/device_vector.h"

#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/common_layers.hpp"

namespace caffe {

template <typename Dtype>
__global__ void kernel_channel_max_sma(const int num,
                                   const int channels,
                                   const int spatial_dim,
                                   const int softmax_size,
                                   const int num_softmaxes,
                                   const Dtype* data,
                                   Dtype* out) {
  CUDA_KERNEL_LOOP(index, num * spatial_dim) {
    int n = index / spatial_dim;
    int s = index % spatial_dim;
    // For each softmax along the canonical axis.
    for( int smi = 0; smi < num_softmaxes; ++smi) {
      Dtype maxval = -FLT_MAX;
      // For each channel within this softmax.
      for (int c_off = 0; c_off < softmax_size; ++c_off) {
        int c = smi * softmax_size + c_off;
        int data_index = (n * channels + c) * spatial_dim + s;
        if (c_off == 0)
        {
          maxval = data[data_index];
        }
        else
        {
          maxval = max(data[data_index], maxval);
        }
      }
      int out_index = s + (n * num_softmaxes + smi) * spatial_dim ;
      out[out_index] = maxval;
    }
  }

}
//CUDA_CHECK(hipMalloc(&gpu_ptr_, size_));

template <typename Dtype>
__global__ void kernel_channel_subtract_sma(const int count,
                                        const int softmax_size,
                                        const int spatial_dim,
                                        const Dtype* channel_max,
                                        Dtype* data,
                                        int* debug_int,
                                        Dtype* debug_float) {
  CUDA_KERNEL_LOOP(index, count) {
    int n = index / softmax_size / spatial_dim;
    int s = index % spatial_dim;

    int softmax_max_index = n * spatial_dim + s;
    Dtype data_before = data[index];
    Dtype channel_max_val = channel_max[softmax_max_index];
    Dtype data_after = data_before - channel_max_val;
    data[index] = data_after;
    if(::isnan(data_after))
    {
      debug_int[0] = softmax_max_index;
      debug_float[0] = data_before;
      debug_float[1] = channel_max_val;
      debug_float[2] = data_after;
    }
    else
    {
      data[index] = data_after;
      if(index == 0)
      {
        debug_int[1] = softmax_max_index;
        debug_float[3] = data_before;
        debug_float[4] = channel_max_val;
        debug_float[5] = data_after;
      }
    }
  }
}

template <typename Dtype>
__global__ void kernel_exp_sma(const int count, const Dtype* data, Dtype* out) {
  CUDA_KERNEL_LOOP(index, count) {
    out[index] = exp(data[index]);
  }
}

template <typename Dtype>
__global__ void kernel_channel_sum_sma(const int num,
                                       const int channels,
                                       const int spatial_dim,
                                       const int softmax_size,
                                       const int num_softmaxes,
                                       const Dtype* data,
                                       Dtype* channel_sum) {
  CUDA_KERNEL_LOOP(index, num * spatial_dim) {
    int n = index / spatial_dim;
    int s = index % spatial_dim;
    // For each softmax along the canonical axis.
    for( int smi = 0; smi < num_softmaxes; ++smi) {
      Dtype sum = 0;
      // For each channel within this softmax.
      for (int c_off = 0; c_off < softmax_size; ++c_off) {
        int c = smi * softmax_size + c_off;
        int data_index = (n * channels + c) * spatial_dim + s;
        sum += data[data_index];
      }
      //int out_index = index*num_softmaxes + smi;
      int out_index = s + (n * num_softmaxes + smi) * spatial_dim ;
      channel_sum[out_index] = sum;
    }
  }
}

// out_probs is only assigned if winner_takes_all is true. Otherwise it is
// ignored. If winner_takes_all, then the out buffer is assigned the binarized
// result.
template <typename Dtype>
__global__ void kernel_channel_div_sma( const int num,
                                    const int channels,
                                    const int spatial_dim,
                                    const int softmax_size,
                                    const int num_softmaxes,
                                    const Dtype* sums,
                                    Dtype* out,
                                    bool winner_take_all,
                                    Dtype* out_probs,
                                    int* debug_int,
                                    Dtype* debug_float) {
  CUDA_KERNEL_LOOP(index, num*spatial_dim) {
    int n = index / spatial_dim;
    int s = index % spatial_dim;
    // For each softmax along the canonical axis.
    for( int smi = 0; smi < num_softmaxes; ++smi) {
      int sum_index = s + (n * num_softmaxes + smi) * spatial_dim ; //index*num_softmaxes + smi;
      Dtype sum = sums[sum_index];

      Dtype largest_prob = -1.0;
      int largest_prob_index = -1;

      // For each channel within this softmax.
      for (int c_off = 0; c_off < softmax_size; ++c_off) {
        int c = smi * softmax_size + c_off;
        int data_index = (n * channels + c) * spatial_dim + s;
        Dtype val_before = out[data_index];
        Dtype val = val_before / sum;

        if (::isnan(val))
        {
          debug_int[0] = data_index;
          debug_float[0] = val_before;
          debug_float[1] = sum;
          debug_float[2] = val;
        }

        if ( winner_take_all) {
          if ( val > largest_prob)
          {
            largest_prob = val;
            largest_prob_index = data_index;
          }
          out_probs[data_index] = val;
        } else {
          if (val < 0.0 ) val = 0.0;
          else if( val > 1.0) val = 1.0;
  //        Dtype val = out[data_index] / sum;
          out[data_index] = val;
        }
      }

      if (winner_take_all)
      {
        for (int c_off = 0; c_off < softmax_size; ++c_off)
        {
          int c = smi * softmax_size + c_off;
          int data_index = (n * channels + c) * spatial_dim + s;
          out[data_index] = ((data_index == largest_prob_index) ?
                1 : 0 );
        }
      }
    }
  }
}

template <typename Dtype>
__global__ void kernel_softmax_dot(const int num,
                                   const int channels,
                                   const int spatial_dim,
                                   const int softmax_size,
                                   const int num_softmaxes,
                                   const Dtype* data_1,
                                   const Dtype* data_2,
                                   Dtype* softmax_dot) {
  CUDA_KERNEL_LOOP(index, num * spatial_dim) {
    int n = index / spatial_dim;
    int s = index % spatial_dim;
    // For each softmax along the canonical axis.
    for( int smi = 0; smi < num_softmaxes; ++smi) {
      Dtype dot = 0;
      // For each channel within this softmax.
      for (int c_off = 0; c_off < softmax_size; ++c_off) {
        int c = smi * softmax_size + c_off;
        int data_index = (n * channels + c) * spatial_dim + s;
        dot += data_1[data_index] * data_2[data_index];
      }
      //int out_index = index*num_softmaxes + smi;
      int out_index = s + (n * num_softmaxes + smi) * spatial_dim ; //index*num_softmaxes + smi;
      softmax_dot[out_index] = dot;
    }
  }
}

template <typename Dtype>
void SoftmaximaLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  int input_count = bottom[0]->count();
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  Dtype* scale_data = scale_.mutable_gpu_data();
  int count = bottom[0]->count();
  int channels = top[0]->shape(softmax_axis_);
  caffe_copy(count, bottom_data, top_data);

//  if( CheckForNanGPU(top[0]->count(), bottom_data) )
//  {
//    LOG(ERROR) << this->layer_param_.name() << "Softmaxima NaN in bottom, A" << std::endl;
//  }

  // NOLINT_NEXT_LINE(whitespace/operators)
  kernel_channel_max_sma<Dtype><<<CAFFE_GET_BLOCKS(outer_num_ * inner_num_),
      CAFFE_CUDA_NUM_THREADS>>>(  outer_num_,
                                  channels,
                                  inner_num_,
                                  softmax_size_,
                                  num_softmaxes_,
                                  top_data,
                                  scale_data);
//  if( CheckForNanGPU(scale_.count(), scale_.gpu_data()) )
//  {
//    LOG(ERROR) << this->layer_param_.name() << "Softmaxima NaN in scale, B"
//               << std::endl;
//  }

  // subtract
  // NOLINT_NEXT_LINE(whitespace/operators)
  kernel_channel_subtract_sma<Dtype><<<CAFFE_GET_BLOCKS(input_count),
      CAFFE_CUDA_NUM_THREADS>>>(input_count, softmax_size_, inner_num_,
      scale_data, top_data, debug_int_.mutable_gpu_data(),
                                debug_float_.mutable_gpu_data());

//  if( CheckForNanGPU(top[0]->count(), top[0]->gpu_data()) )
//  {
//    std::cout << "NaN index, before, max, after = " << debug_int_.cpu_data()[0]
//                 << "," << debug_float_.cpu_data()[0] << ","
//                    << debug_float_.cpu_data()[1] << ","
//                       << debug_float_.cpu_data()[2] << std::endl;
//    std::cout << "Good index, before, max, after = " << debug_int_.cpu_data()[1]
//                 << "," << debug_float_.cpu_data()[3] << ","
//                    << debug_float_.cpu_data()[4] << ","
//                       << debug_float_.cpu_data()[5] << std::endl;
//    LOG(ERROR) << this->layer_param_.name() << "Softmaxima NaN C." << std::endl;
//  }

  // NOLINT_NEXT_LINE(whitespace/operators)
  kernel_exp_sma<Dtype><<<CAFFE_GET_BLOCKS(input_count), CAFFE_CUDA_NUM_THREADS>>>(
      input_count, top_data, top_data);

//  if( CheckForNanGPU(top[0]->count(), top[0]->gpu_data()) )
//  {
//    LOG(ERROR) << this->layer_param_.name() << "Softmaxima NaN D." << std::endl;
//  }

  // NOLINT_NEXT_LINE(whitespace/operators)
  kernel_channel_sum_sma<Dtype><<<CAFFE_GET_BLOCKS(outer_num_ * inner_num_),
      CAFFE_CUDA_NUM_THREADS>>>(outer_num_,
                                channels,
                                inner_num_,
                                softmax_size_,
                                num_softmaxes_,
                                top_data,
                                scale_data);

//  if( CheckForNanGPU(scale_.count(), scale_.gpu_data()) )
//  {
//    LOG(ERROR) << this->layer_param_.name() << "Softmaxima NaN E." << std::endl;
//  }

  Dtype* output_probs_buffer = WinnerTakeAll() ?
        this->output_probs_.mutable_gpu_data() : 0;

  // NOLINT_NEXT_LINE(whitespace/operators)
  kernel_channel_div_sma<Dtype><<<CAFFE_GET_BLOCKS(outer_num_*inner_num_),
      CAFFE_CUDA_NUM_THREADS>>>(outer_num_,
                                channels,
                                inner_num_,
                                softmax_size_,
                                num_softmaxes_,
                                scale_data,
                                top_data,
                                WinnerTakeAll(),
                                output_probs_buffer,
                                debug_int_.mutable_gpu_data(),
                                debug_float_.mutable_gpu_data());

//  if( CheckForNanGPU(top[0]->count(), top[0]->gpu_data()))
//  {
//    std::cout << "NaN index, before, sum, after = " << debug_int_.cpu_data()[0]
//                 << "," << debug_float_.cpu_data()[0] << ","
//                    << debug_float_.cpu_data()[1] << ","
//                       << debug_float_.cpu_data()[2] << std::endl;
//    LOG(ERROR) << this->layer_param_.name() << " NaN F." << std::endl;
//  }

  Dtype test_val;
  if( CheckForOutOfRangeGPU(top[0]->count(), top[0]->gpu_data(),
                            test_val))
  {
    LOG(FATAL) << "Found softmaxima output not between 0 and 1: "
               << test_val << ", in top of layer " << this->layer_param_.name();
  }


//  CheckForNanGPU("softmaxima5", "top", *top[0]);
//  if( WinnerTakeAll())
//  {
//    CheckForNanGPU("softmaxima6", "output_probs", this->output_probs_);
//  }
}

template <typename Dtype>
void SoftmaximaLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  const Dtype* top_diff = top[0]->gpu_diff();

  // Use mean-field activations for the backprop if WinnerTakeAll.
  const Dtype* top_data = WinnerTakeAll() ? output_probs_.gpu_data() :
                                            top[0]->gpu_data();
  Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
  Dtype* scale_data = scale_.mutable_gpu_data();
  int count = top[0]->count();
  int channels = top[0]->shape(softmax_axis_);
  caffe_copy(count, top_diff, bottom_diff);
  // Compute inner1d(top_diff, top_data) and subtract them from the bottom diff.
  // NOLINT_NEXT_LINE(whitespace/operators)
  kernel_softmax_dot<Dtype><<<CAFFE_GET_BLOCKS(outer_num_ * inner_num_),
      CAFFE_CUDA_NUM_THREADS>>>(outer_num_,
                                channels,
                                inner_num_,
                                softmax_size_,
                                num_softmaxes_,
                                top_diff,
                                top_data,
                                scale_data);
  // NOLINT_NEXT_LINE(whitespace/operators)
  kernel_channel_subtract_sma<Dtype><<<CAFFE_GET_BLOCKS(count),
      CAFFE_CUDA_NUM_THREADS>>>(count, softmax_size_, inner_num_,
      scale_data, bottom_diff, debug_int_.mutable_gpu_data(),
                                debug_float_.mutable_gpu_data());
  // elementwise multiplication
  caffe_gpu_mul<Dtype>(top[0]->count(), bottom_diff, top_data, bottom_diff);
}

INSTANTIATE_LAYER_GPU_FUNCS(SoftmaximaLayer);


}  // namespace caffe
